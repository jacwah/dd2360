
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdint.h>


// 40 GB memory
// sizeof(Particle) = 4*3*2 = 24 bytes
// 40 GB / 24 B = 1.6 G particles
#define NUM_PARTICLES 1.6e9
#define NUM_ITERATIONS 10


struct Particle {
    float3 position;
    float3 velocity;
};

__host__ __device__ void update_particle(Particle* particle) { 
        particle->velocity.x = particle->position.x;
        particle->velocity.y = particle->position.y;
        particle->velocity.z = particle->position.z;
        particle->position.x += particle->velocity.x; 
        particle->position.y += particle->velocity.y;
        particle->position.z += particle->velocity.z;
}

__global__ void update_kernel(Particle* particles, const uint64_t n) { 
    uint64_t i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) {    
        update_particle(particles+i);
    }
}

void update(Particle* particles, const uint64_t n) {
    for (uint64_t i = 0; i < n; i++) {
        update_particle(particles+i);
    }
}

    
void initialize_array(Particle *particles, const uint64_t n) {
    for (uint64_t i = 0; i < n; i++) {
        particles[i].position.x = (float)rand()/RAND_MAX; 
        particles[i].position.y = (float)rand()/RAND_MAX; 
        particles[i].position.z = (float)rand()/RAND_MAX; 
        particles[i].velocity = {}; 
    }
}

void compare(const Particle *p_cpu, const Particle *p_gpu, const uint64_t n) {
    for (uint64_t i = 0; i < n; i++) {
        if (abs(p_cpu[i].position.x - p_gpu[i].position.x) > 1e6 ||
        abs(p_cpu[i].position.y - p_gpu[i].position.y) > 1e6 || 
        abs(p_cpu[i].position.z - p_gpu[i].position.z) > 1e6 ||
        abs(p_cpu[i].velocity.x - p_gpu[i].velocity.x) > 1e6 ||
        abs(p_cpu[i].velocity.y - p_gpu[i].velocity.y) > 1e6 ||
        abs(p_cpu[i].velocity.z - p_gpu[i].velocity.z) > 1e6) {
            printf("Result not equal\n");
            return;
        }
    }
    printf("Comparison OK\n");
}


int main() {
    Particle *particles_d = NULL;
    uint64_t arraySize = sizeof(Particle)*NUM_PARTICLES;    
    timeval gpu_t1;   
    timeval gpu_t2;   
    const uint64_t BLOCK_SIZE = 64;
    Particle *particles = NULL;
    
    hipHostMalloc(&particles, arraySize, hipHostMallocDefault);

    initialize_array(particles, NUM_PARTICLES);
    
    hipMalloc(&particles_d, arraySize);
    
    gettimeofday(&gpu_t1, NULL);

    for (uint64_t i = 0; i < NUM_ITERATIONS; i++) {
        hipMemcpy(particles_d, particles, arraySize, hipMemcpyDefault);

        update_kernel<<<(NUM_PARTICLES + BLOCK_SIZE-1)/BLOCK_SIZE,BLOCK_SIZE>>>(particles_d, NUM_PARTICLES);

        hipMemcpy(particles, particles_d, arraySize, hipMemcpyDefault);
    }

    gettimeofday(&gpu_t2, NULL);
    printf("GPU time: %e seconds \n", ((gpu_t2.tv_sec + gpu_t2.tv_usec/1e6) - (gpu_t1.tv_sec + gpu_t1.tv_usec/1e6)));
    
    hipHostFree(particles);
    hipFree(particles_d);
}
