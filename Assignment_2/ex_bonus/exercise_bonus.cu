
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SEED     921
#define NUM_ITER 100000000

__global__ void calc_prob(const int iterations, unsigned long long *counts) {
    unsigned long long count = 0;
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    extern __shared__ hiprandState state[];
    
    hiprand_init(SEED, idx, 0, &state[threadIdx.x]); 
    for (int iter = 0; iter < iterations; iter++)
    {
        double x, y, z;

        // Generate random (X,Y) points
        x = hiprand_uniform(&state[threadIdx.x]);
        y = hiprand_uniform(&state[threadIdx.x]);
        z = (x*x) + (y*y);

        // Check if point is in unit circle
        if (z <= 1.0) {
            count++;
        }
    }

    atomicAdd_block(&counts[blockIdx.x], count);
}


int main(int argc, char* argv[])
{
    //double pi;
    //int blocks, iterations;
    //srand(SEED); // Important: Multiply SEED by "rank" when you introduce MPI!
    
    for(int i = 16; i <= 256; i*=2) {
        unsigned long long int count = 0;
        unsigned long long *counts = NULL;
        
        int blocks = (640 + (i - 1))/i;
        printf("Blocks: %i  Threads per block: %i\n", blocks, i);
        
        hipMalloc(&counts, sizeof(unsigned long long)*blocks);
        hipMemset(counts, 0, sizeof(unsigned long long)*blocks);
        unsigned long long *counts_h = (unsigned long long*)malloc(sizeof(unsigned long long)*blocks);
        
        
        int iterations = (NUM_ITER + (i*blocks - 1))/ (i*blocks);
        printf("Total itterations: %i\n", iterations);
        
        calc_prob<<<blocks, i, i*sizeof(hiprandState)>>>(iterations, counts);
        
        hipMemcpy(counts_h, counts, sizeof(unsigned long long) * blocks, hipMemcpyDefault);
        
        for (int j = 0; j < blocks; j++) {
            // printf("Counts_h[%d] = %llu\n", j, counts_h[j]);
            count += counts_h[j];
        }
        
        // Estimate Pi and display the result
        double pi = ((double)count / (double)(iterations*i*blocks)) * 4.0;
    
        printf("The result is %f\n", pi);

        hipFree(counts);
        free(counts_h);   
    }
    
    
    return 0;
}
