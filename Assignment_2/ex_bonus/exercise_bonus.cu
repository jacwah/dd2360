
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SEED     921
#define NUM_ITER 1e10

#ifndef real
#define real double
#endif

__global__ void calc_prob(const long long iterations, unsigned long long *counts) {
    unsigned long long count = 0;
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    extern __shared__ hiprandState state[];
    
    hiprand_init(SEED, idx, 0, &state[threadIdx.x]); 
    for (long long iter = 0; iter < iterations; iter++)
    {
        real x, y, z;

        // Generate random (X,Y) points
        x = hiprand_uniform(&state[threadIdx.x]);
        y = hiprand_uniform(&state[threadIdx.x]);
        z = (x*x) + (y*y);

        // Check if point is in unit circle
        if (z <= ((real)1.0)) {
            count++;
        }
    }

#if __CUDA_ARCH__ >= 600
    atomicAdd_block(&counts[blockIdx.x], count);
#else
    atomicAdd(&counts[blockIdx.x], count);
#endif
}


int main(int argc, char* argv[])
{
    //double pi;
    //int blocks, iterations;
    //srand(SEED); // Important: Multiply SEED by "rank" when you introduce MPI!
    
    for(int i = 16; i <= 256; i*=2) {
        unsigned long long int count = 0;
        unsigned long long *counts = NULL;
        
        int blocks = (640 + (i - 1))/i;
        printf("Blocks: %i  Threads per block: %i\n", blocks, i);
        
        hipMalloc(&counts, sizeof(unsigned long long)*blocks);
        hipMemset(counts, 0, sizeof(unsigned long long)*blocks);
        unsigned long long *counts_h = (unsigned long long*)malloc(sizeof(unsigned long long)*blocks);
        
        
        long long iterations = (NUM_ITER + (i*blocks - 1))/ (i*blocks);
        printf("Total itterations: %lld\n", iterations);
        
        calc_prob<<<blocks, i, i*sizeof(hiprandState)>>>(iterations, counts);
        
        hipMemcpy(counts_h, counts, sizeof(unsigned long long) * blocks, hipMemcpyDefault);
        
        for (int j = 0; j < blocks; j++) {
            // printf("Counts_h[%d] = %llu\n", j, counts_h[j]);
            count += counts_h[j];
        }
        
        // Estimate Pi and display the result
        double pi = ((double)count / (double)(iterations*i*blocks)) * 4.0;
    
        printf("The result is %f\n", pi);

        hipFree(counts);
        free(counts_h);   
    }
    
    
    return 0;
}
